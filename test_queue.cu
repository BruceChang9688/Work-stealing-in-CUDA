#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda/c_Queue.h"

__global__ void test_queue_in_shared_memory(int capacity)
{
    extern __shared__ QueueSlot slots[];
    __shared__ int queueParam[4];
    
    if(threadIdx.x == 0)
    {
        queueParam[CAPACITY] = capacity;
        queueParam[REARIDX] = REARIDX;
        queueParam[FRONTIDX] = FRONTIDX;
        queueParam[NUMWAITINGTASKS] = 0;
    }
    __syncthreads();

    Queue queue(slots, queueParam);
    
    int globalThreadId = blockDim.x*blockIdx.x + threadIdx.x;
    
    Task task;
    task.ray = Ray(Point(globalThreadId, globalThreadId, globalThreadId), Vector3D(1, 1, 1));

    QueueSlot slot;
    slot.pixelIndex = globalThreadId;
    slot.task = task;

    queue.enqueue(slot);

    int* paramThread = queue.param();
    //printf("Global threadId: %d, rearIdx: %d, numWaitingTasks: %d\n", globalThreadId, paramThread[1], paramThread[3]);

}


int main(int argc, char** argv)
{

    int blocks = 1;
    int threads = 33;

    test_queue_in_shared_memory<<<blocks, threads, (threads + 1)*sizeof(QueueSlot)>>>(threads + 1);
    hipDeviceSynchronize();
}